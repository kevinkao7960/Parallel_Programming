
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#define BYTE unsigned char
#define BLOCKSIZE 16

int pic_len;

void printBytes(BYTE b[], int len) {
    int i;
    for (i=0; i<len; i++)
    printf("%d ", b[i]);
    printf("\n");
}

/******************************************************************************/

// The following lookup tables and functions are for internal use only!
BYTE AES_Sbox[] = {99,124,119,123,242,107,111,197,48,1,103,43,254,215,171,
    118,202,130,201,125,250,89,71,240,173,212,162,175,156,164,114,192,183,253,
    147,38,54,63,247,204,52,165,229,241,113,216,49,21,4,199,35,195,24,150,5,154,
    7,18,128,226,235,39,178,117,9,131,44,26,27,110,90,160,82,59,214,179,41,227,
    47,132,83,209,0,237,32,252,177,91,106,203,190,57,74,76,88,207,208,239,170,
    251,67,77,51,133,69,249,2,127,80,60,159,168,81,163,64,143,146,157,56,245,
    188,182,218,33,16,255,243,210,205,12,19,236,95,151,68,23,196,167,126,61,
    100,93,25,115,96,129,79,220,34,42,144,136,70,238,184,20,222,94,11,219,224,
    50,58,10,73,6,36,92,194,211,172,98,145,149,228,121,231,200,55,109,141,213,
    78,169,108,86,244,234,101,122,174,8,186,120,37,46,28,166,180,198,232,221,
    116,31,75,189,139,138,112,62,181,102,72,3,246,14,97,53,87,185,134,193,29,
    158,225,248,152,17,105,217,142,148,155,30,135,233,206,85,40,223,140,161,
    137,13,191,230,66,104,65,153,45,15,176,84,187,22};

BYTE AES_ShiftRowTab[] = {0,5,10,15,4,9,14,3,8,13,2,7,12,1,6,11};

BYTE AES_Sbox_Inv[256];
BYTE AES_ShiftRowTab_Inv[16];
BYTE AES_xtime[256];

__device__ void AES_SubBytes(BYTE state[], BYTE sbox[]) {
    int i;
    for(i = 0; i < 16; i++)
        state[i] = sbox[state[i]];
}

__device__ void AES_AddRoundKey(BYTE state[], BYTE rkey[]) {
    int i;
    for(i = 0; i < 16; i++)
        state[i] ^= rkey[i];
}

__device__ void AES_ShiftRows(BYTE state[], BYTE shifttab[]) {
    BYTE h[16];
    memcpy(h, state, 16);
    int i;
    for(i = 0; i < 16; i++)
        state[i] = h[shifttab[i]];
}

__device__ void AES_MixColumns(BYTE state[], BYTE aes_xtime[]) {
    int i;
    for(i = 0; i < 16; i += 4) {
        BYTE s0 = state[i + 0], s1 = state[i + 1];
        BYTE s2 = state[i + 2], s3 = state[i + 3];
        BYTE h = s0 ^ s1 ^ s2 ^ s3;
        state[i + 0] ^= h ^ aes_xtime[s0 ^ s1];
        state[i + 1] ^= h ^ aes_xtime[s1 ^ s2];
        state[i + 2] ^= h ^ aes_xtime[s2 ^ s3];
        state[i + 3] ^= h ^ aes_xtime[s3 ^ s0];
    }
}

__device__ void AES_MixColumns_Inv(BYTE state[], BYTE aes_xtime[]) {
    int i;
    for(i = 0; i < 16; i += 4) {
        BYTE s0 = state[i + 0], s1 = state[i + 1];
        BYTE s2 = state[i + 2], s3 = state[i + 3];
        BYTE h = s0 ^ s1 ^ s2 ^ s3;
        BYTE xh = aes_xtime[h];
        BYTE h1 = aes_xtime[aes_xtime[xh ^ s0 ^ s2]] ^ h;
        BYTE h2 = aes_xtime[aes_xtime[xh ^ s1 ^ s3]] ^ h;
        state[i + 0] ^= h1 ^ aes_xtime[s0 ^ s1];
        state[i + 1] ^= h2 ^ aes_xtime[s1 ^ s2];
        state[i + 2] ^= h1 ^ aes_xtime[s2 ^ s3];
        state[i + 3] ^= h2 ^ aes_xtime[s3 ^ s0];
    }
}

// AES_Init: initialize the tables needed at runtime.
// Call this function before the (first) key expansion.
void AES_Init() {
    int i;
    for(i = 0; i < 256; i++)
        AES_Sbox_Inv[AES_Sbox[i]] = i;

    for(i = 0; i < 16; i++)
        AES_ShiftRowTab_Inv[AES_ShiftRowTab[i]] = i;

    for(i = 0; i < 128; i++) {
        AES_xtime[i] = i << 1;
        AES_xtime[128 + i] = (i << 1) ^ 0x1b;
    }
}

// AES_Done: release memory reserved by AES_Init.
// Call this function after the last encryption/decryption operation.
void AES_Done() {}

/* AES_ExpandKey: expand a cipher key. Depending on the desired encryption
strength of 128, 192 or 256 bits 'key' has to be a byte array of length
16, 24 or 32, respectively. The key expansion is done "in place", meaning
that the array 'key' is modified.
*/
int AES_ExpandKey(BYTE key[], int keyLen) {
    int kl = keyLen, ks, Rcon = 1, i, j;
    BYTE temp[4], temp2[4];
    switch (kl) {
        case 16: ks = 16 * (10 + 1); break;
        case 24: ks = 16 * (12 + 1); break;
        case 32: ks = 16 * (14 + 1); break;
        default:
            printf("AES_ExpandKey: Only key lengths of 16, 24 or 32 bytes allowed!");
    }
    for(i = kl; i < ks; i += 4) {
        memcpy(temp, &key[i-4], 4);
        if (i % kl == 0) {
            temp2[0] = AES_Sbox[temp[1]] ^ Rcon;
            temp2[1] = AES_Sbox[temp[2]];
            temp2[2] = AES_Sbox[temp[3]];
            temp2[3] = AES_Sbox[temp[0]];
            memcpy(temp, temp2, 4);
            if ((Rcon <<= 1) >= 256)
                Rcon ^= 0x11b;
        }
        else if ((kl > 24) && (i % kl == 16)) {
            temp2[0] = AES_Sbox[temp[0]];
            temp2[1] = AES_Sbox[temp[1]];
            temp2[2] = AES_Sbox[temp[2]];
            temp2[3] = AES_Sbox[temp[3]];
            memcpy(temp, temp2, 4);
        }
        for(j = 0; j < 4; j++)
            key[i + j] = key[i + j - kl] ^ temp[j];
    }
    return ks;
}

// AES_Encrypt: encrypt the 16 byte array 'block' with the previously expanded key 'key'.
__global__ void AES_Encrypt(BYTE block[], BYTE key[], int keyLen, BYTE aes_sbox[], BYTE aes_shiftrowtab[], BYTE aes_xtime[]) {
    int l = keyLen, i;
    int j = threadIdx.x;
    int k = j + blockIdx.x * 16;

    BYTE device_block[16];
    for( int idx = k*16; idx < k*16+16; idx++){
        device_block[idx - k*16] = block[idx];
    }

    // printBytes(block, 16);
    AES_AddRoundKey(device_block, &key[0]);
    for(i = 16; i < l - 16; i += 16) {
        AES_SubBytes(device_block, aes_sbox);
        AES_ShiftRows(device_block, aes_shiftrowtab);
        AES_MixColumns(device_block, aes_xtime);
        AES_AddRoundKey(device_block, &key[i]);
    }
    AES_SubBytes(device_block, aes_sbox);
    AES_ShiftRows(device_block, aes_shiftrowtab);
    AES_AddRoundKey(device_block, &key[i]);

    for( int idx = k*16; idx < k*16+16; idx++ ){
        block[idx] = device_block[idx - k*16];
    }
}

// AES_Decrypt: decrypt the 16 byte array 'block' with the previously expanded key 'key'.
__global__ void AES_Decrypt(BYTE block[], BYTE key[], int keyLen, BYTE aes_xtime[],BYTE aes_shiftrowtab_inv[], BYTE aes_sbox_inv[]) {
    int l = keyLen, i;
    AES_AddRoundKey(block, &key[l - 16]);
    AES_ShiftRows(block, aes_shiftrowtab_inv);
    AES_SubBytes(block, aes_sbox_inv);
    for(i = l - 32; i >= 16; i -= 16) {
        AES_AddRoundKey(block, &key[i]);
        AES_MixColumns_Inv(block, aes_xtime);
        AES_ShiftRows(block, aes_shiftrowtab_inv);
        AES_SubBytes(block, aes_sbox_inv);
    }
    AES_AddRoundKey(block, &key[0]);
}

BYTE* readFile(char *filename){
    FILE *file;
    BYTE *buffer;
    unsigned long fileLen;

    // Open file
    file = fopen(filename, "rb");
    if(!file){
        fprintf(stderr, "Unable to open file %s", filename);
        return 0;
    }

    // Get file lengths
    fseek(file, 0, SEEK_END);
    fileLen = ftell(file);
    fseek(file, 0, SEEK_SET);

    // Allocate memory
    buffer = (BYTE*)malloc(fileLen + 1);
    if(!buffer){
        fprintf(stderr, "Memory error!");
        fclose(file);
        return 0;
    }

    fread(buffer, fileLen, sizeof(BYTE), file);
    fclose(file);

    pic_len = fileLen;

    return buffer;
}

// ===================== test ============================================
int main() {
    // int i;
    AES_Init();

    // BYTE block[16];
    // for(i = 0; i < 16; i++)
    //     block[i] = 0x11 * i;

    BYTE *pic;
    pic = readFile("minions.jpg");

    if( !pic ){
        fprintf(stderr, "Memory creation error");
    }
    else{
        // delete jpg header (11 bytes)
        for( int i = 11; i < pic_len; i++ ){
            pic[i-11] = pic[i];
        }
        printf("%d\n", pic_len);
        // delete the last header (2 bytes)
        pic_len -= 11;
        pic[pic_len-1] = 0;
        pic[pic_len-2] = 0;
        pic_len -= 2;
        pic = (BYTE*)realloc(pic, pic_len);
        printf("%d\n", pic_len);

        // align the last block
        // if( pic_len % 16 ){
        //     int align_num = 16 - pic_len % 16;
        //     pic_len += align_num;
        //     pic = (BYTE*)realloc(pic, pic_len);
        //     for( int j = 0; j < align_num; j++ ){
        //         pic[pic_len - j - 1] = 1;
        //     }
        // }



        // allocate the cuda device space
        BYTE *pic_d, *key_d, *AES_Sbox_d, *AES_ShiftRowTab_d, *AES_Sbox_Inv_d, *AES_ShiftRowTab_Inv_d, *AES_xtime_d, *encrypt_result;
        BYTE key[16 * (14 + 1)];

        int keyLen = 32, maxKeyLen=16 * (14 + 1);
        for( int j = 0; j < keyLen; j++ ){
            key[j] = j;
        }
        int expandKeyLen = AES_ExpandKey(key, keyLen);

        hipMalloc( &pic_d, pic_len*sizeof(BYTE) );
        hipMemcpy( pic_d, pic, pic_len*sizeof(BYTE), hipMemcpyHostToDevice);
        hipMalloc( &key_d, expandKeyLen);
        hipMemcpy( key_d, key, expandKeyLen*sizeof(BYTE), hipMemcpyHostToDevice);

        hipMalloc( &AES_Sbox_d, sizeof(AES_Sbox)/sizeof(BYTE));
        hipMemcpy( AES_Sbox_d, AES_Sbox, sizeof(AES_Sbox)/sizeof(BYTE), hipMemcpyHostToDevice);
        hipMalloc( &AES_ShiftRowTab_d, sizeof(AES_ShiftRowTab)/sizeof(BYTE));
        hipMemcpy( AES_ShiftRowTab_d, AES_ShiftRowTab, sizeof(AES_ShiftRowTab)/sizeof(BYTE), hipMemcpyHostToDevice);
        hipMalloc( &AES_Sbox_Inv_d, sizeof(AES_Sbox_Inv)/sizeof(BYTE));
        hipMemcpy( AES_Sbox_Inv_d, AES_Sbox_Inv, sizeof(AES_Sbox_Inv)/sizeof(BYTE), hipMemcpyHostToDevice);
        hipMalloc( &AES_ShiftRowTab_Inv_d, sizeof(AES_ShiftRowTab_Inv)/sizeof(BYTE));
        hipMemcpy( AES_ShiftRowTab_Inv_d, AES_ShiftRowTab_Inv, sizeof(AES_ShiftRowTab_Inv)/sizeof(BYTE), hipMemcpyHostToDevice);
        hipMalloc( &AES_xtime_d, sizeof(AES_xtime)/sizeof(BYTE));
        hipMemcpy( AES_xtime_d, AES_xtime, sizeof(AES_xtime)/sizeof(BYTE), hipMemcpyHostToDevice);


        /**
         * Block Size: 16 thread
         * Grid Size: BlockNum, 1
         **/
        int blockNum;
        if( pic_len % 16 ){
            blockNum = 1 + pic_len / 16;
        }
        else{
            blockNum = pic_len / 16;
        }

        dim3 dimGrid(blockNum, 1);
        dim3 dimBlock(16, 1);

        AES_Encrypt<<<dimGrid, dimBlock>>>(pic_d, key_d, expandKeyLen, AES_Sbox_d, AES_ShiftRowTab_d, AES_xtime_d);

        /* get the encrypt result */
        encrypt_result = (BYTE*)malloc(sizeof(BYTE)*pic_len);
        hipMemcpy( encrypt_result, pic_d, pic_len*sizeof(BYTE), hipMemcpyDeviceToHost);
        printf("%d\n", pic_len);
        int j = 0;
        while( j < pic_len){
            printf("%02x ", (BYTE)encrypt_result[j]);
            j++;
            if( !(j%16) ) printf("\n");
        }
        hipFree(pic_d);
        hipFree(key_d);
        hipFree(AES_Sbox_d);
        hipFree(AES_ShiftRowTab_d);
        hipFree(AES_Sbox_Inv_d);
        hipFree(AES_ShiftRowTab_Inv_d);
        hipFree(AES_xtime_d);
    }

    // printf("原始訊息："); printBytes(block, 16);

    // BYTE key[16 * (14 + 1)];
    // int keyLen = 32, maxKeyLen=16 * (14 + 1), blockLen = 16;
    // for(i = 0; i < keyLen; i++)
    //     key[i] = i;

    // printf("原始金鑰："); printBytes(key, keyLen);

    // int expandKeyLen = AES_ExpandKey(key, keyLen);

    // printf("展開金鑰："); printBytes(key, expandKeyLen);

    // AES_Encrypt(block, key, expandKeyLen);

    // printf("加密完後："); printBytes(block, blockLen);

    // AES_Decrypt(block, key, expandKeyLen, aes_xtime_d);

    // printf("解密完後："); printBytes(block, blockLen);

    // AES_Done();
}
